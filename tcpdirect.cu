
#include <hip/hip_runtime.h>
#ifdef WITH_TCPDIRECT
#include <cuda.h>
#include <cuda_runtime.h>

#include <asm-generic/errno-base.h>
#include <asm-generic/socket.h>
#include <errno.h>
#include <fcntl.h>
#include <linux/if.h>
#include <stdlib.h>
#include <sys/ioctl.h>
#include <sys/mman.h>
#include <sys/socket.h>
#include <unistd.h>
#include <linux/dma-buf.h>

#include <memory>
#include <cstdint>
#include <cstdio>
#include <string>

#include "tcpdirect.h"
#include "logging.h"
#include "flow.h"
#include "thread.h"

#define MIN_RX_BUFFER_TOTAL_SIZE (1 << 28)
#define GPUMEM_ALIGNMENT (1UL << 21)
#define GPUMEM_MINSZ 0x400000
#define PAGE_SHIFT (12)
#define PAGE_SIZE (1 << PAGE_SHIFT)

#define multiplier (1 << 16)

#define SO_DEVMEM_OFFSET 99
#define SCM_DEVMEM_OFFSET SO_DEVMEM_OFFSET

#define TEST_PREFIX "ncdevmem"
#define NUM_PAGES 16000

/* missing definitions in mman-linux.h */
#ifndef MFD_ALLOW_SEALING
#define MFD_ALLOW_SEALING 2U
#endif

/* GRTE libraries from google3 already define the following */
#ifndef F_SEAL_SHRINK
#define F_SEAL_SHRINK 2U
#endif
#ifndef F_ADD_SEALS
#define F_ADD_SEALS 1033U
#endif
#ifndef F_GET_SEALS
#define F_GET_SEALS 1034U
#endif

#define MSG_SOCK_DEVMEM 0x2000000
#define SO_DEVMEM_DONTNEED 97
#define SO_DEVMEM_HEADER 98
#define SCM_DEVMEM_HEADER SO_DEVMEM_HEADER
#define SO_DEVMEM_OFFSET 99
#define SCM_DEVMEM_OFFSET SO_DEVMEM_OFFSET

struct dma_buf_create_pages_info {
  __u64 pci_bdf[3];
  __s32 dma_buf_fd;
  __s32 create_page_pool;
};

struct dma_buf_pages_bind_rx_queue {
  char ifname[IFNAMSIZ];
  __u32 rxq_idx;
};

#define DMA_BUF_CREATE_PAGES \
  _IOW(DMA_BUF_BASE, 2, struct dma_buf_create_pages_info)

#define DMA_BUF_PAGES_BIND_RX \
  _IOW(DMA_BUF_BASE, 3, struct dma_buf_pages_bind_rx_queue)

// devmemvec represents a fragment of payload that is received on the socket.
struct devmemvec {
  // frag_offset is the offset in the registered memory.
  __u32 frag_offset;
  // frag size is the size of the payload.
  __u32 frag_size;
  // frag_token is an identifier for this fragment and it can be used to return
  // the memory back to kernel.
  __u32 frag_token;
};

// devmemtoken represents a range of tokens. It is used to return the fragment
// memory back to the kernel.
struct devmemtoken {
  __u32 token_start;
  __u32 token_count;
};

struct udmabuf_create {
  uint32_t memfd;
  uint32_t flags;
  uint64_t offset;
  uint64_t size;
};
#define UDMABUF_CREATE _IOW('u', 0x42, struct udmabuf_create)

int memfd_create(const char *name, unsigned int flags)
{
	return syscall(__NR_memfd_create, name, flags);
}

int tcpdirect_setup_socket(int socket) {
  const int one = 1;
  if (setsockopt(socket, SOL_SOCKET,
                 SO_REUSEADDR | SO_REUSEPORT | SO_ZEROCOPY,
                 &one,
                 sizeof(one))) {
    perror("tcpdirect_setup_socket");
    exit(EXIT_FAILURE);
  }

  return 0;
}

int get_gpumem_dmabuf_pages_fd(const std::string& gpu_pci_addr,
                               const std::string& nic_pci_addr, void* gpu_mem,
                               size_t gpu_mem_sz, int* dma_buf_fd, bool is_client) {
  int err, ret;

  cuMemGetHandleForAddressRange((void*)dma_buf_fd, (CUdeviceptr)gpu_mem,
                                gpu_mem_sz, CU_MEM_RANGE_HANDLE_TYPE_DMA_BUF_FD,
                                0);

  if (*dma_buf_fd < 0) {
    perror("cuMemGetHandleForAddressRange() failed!: ");
    exit(EXIT_FAILURE);
  }

  printf("Registered dmabuf region 0x%p of %lu Bytes\n",
      gpu_mem, gpu_mem_sz);

  struct dma_buf_create_pages_info frags_create_info;
  frags_create_info.dma_buf_fd = *dma_buf_fd;
  frags_create_info.create_page_pool = is_client ? 0 : 1;

  uint16_t pci_bdf[3];
  ret = sscanf(nic_pci_addr.c_str(), "0000:%hx:%hx.%hx", &pci_bdf[0],
               &pci_bdf[1], &pci_bdf[2]);
  frags_create_info.pci_bdf[0] = pci_bdf[0];
  frags_create_info.pci_bdf[1] = pci_bdf[1];
  frags_create_info.pci_bdf[2] = pci_bdf[2];
  if (ret != 3) {
    err = -EINVAL;
    goto err_close_dmabuf;
  }

  ret = ioctl(*dma_buf_fd, DMA_BUF_CREATE_PAGES, &frags_create_info);
  if (ret < 0) {
    perror("Error getting dma_buf frags: ");
    err = -EIO;
    goto err_close_dmabuf;
  }
  return ret;

err_close_dmabuf:
  close(*dma_buf_fd);
  return err;
}

int tcpdirect_cuda_setup_alloc(const struct options *opts, void **f_mbuf, struct thread *t)
{
  bool is_client = opts->client;
  int ret;
  void *gpu_tx_mem_;
  int gpu_mem_fd_;
  int dma_buf_fd_;
  std::unique_ptr<char[]> buf_;
  struct tcpdirect_cuda_mbuf *tmbuf;
  const char *gpu_pci_addr = opts->tcpd_gpu_pci_addr;  // "0000:04:00.0"
  const char *nic_pci_addr = opts->tcpd_nic_pci_addr;  // "0000:06:00.0"
  size_t message_size_ = 4096000; // TODO param this
  size_t alloc_size = opts->tcpdirect_phys_len;  // std::max(message_size_, (unsigned long)GPUMEM_MINSZ)

  tmbuf =
    (struct tcpdirect_cuda_mbuf *)calloc(1, sizeof(struct tcpdirect_udma_mbuf));
  if (!tmbuf) {
    exit(EXIT_FAILURE);
  }

  if (alloc_size % GPUMEM_ALIGNMENT != 0) {
    alloc_size += GPUMEM_ALIGNMENT - (alloc_size % GPUMEM_ALIGNMENT);
  }

  ret = cudaSetDevice(opts->tcpdirect_gpu_idx);
  if (ret != 0) {
    printf("cudaSetDevice failed: index %i", opts->tcpdirect_gpu_idx);
    exit(70);
  }

  cudaMalloc(&gpu_tx_mem_, alloc_size);
  unsigned int flag = 1;
  cuPointerSetAttribute(&flag,
                        CU_POINTER_ATTRIBUTE_SYNC_MEMOPS,
                        (CUdeviceptr)gpu_tx_mem_);
  buf_.reset(new char[message_size_]);

  gpu_mem_fd_ = get_gpumem_dmabuf_pages_fd(gpu_pci_addr, nic_pci_addr,
                                           gpu_tx_mem_, alloc_size,
                                           &dma_buf_fd_, is_client);

  if (gpu_mem_fd_ < 0) {
    printf("get_gpumem_dmabuf_pages_fd() failed!: ");
    exit(71);
  }

  if (!is_client) {
    /* TODO hardcoded num_queues */
    int num_queues = 15;
    struct dma_buf_pages_bind_rx_queue bind_cmd;

    strcpy(bind_cmd.ifname, opts->tcpdirect_link_name);
    bind_cmd.rxq_idx = num_queues;

    ret = ioctl(gpu_mem_fd_, DMA_BUF_PAGES_BIND_RX, &bind_cmd);
    if (ret < 0) {
      printf("%s: [FAIL, bind fail queue=%d]\n", TEST_PREFIX,
            num_queues);
      exit(78);
    }
  }

  *f_mbuf = tmbuf;
  tmbuf->gpu_mem_fd_ = gpu_mem_fd_;
  tmbuf->dma_buf_fd_ = dma_buf_fd_;
  tmbuf->gpu_tx_mem_ = gpu_tx_mem_;
  return 0;
}

int udmabuf_setup_alloc(const struct options *opts, void **f_mbuf) {
  bool is_client = opts->client;
  int devfd;
  int memfd;
  int buf;
  int buf_pages;
  int ret;
  size_t size = opts->tcpdirect_phys_len;

  struct tcpdirect_udma_mbuf *tmbuf;
  struct dma_buf_create_pages_info pages_create_info;
  struct udmabuf_create create;

  if (f_mbuf == NULL) return ENOMEM;

  if (*f_mbuf) return 0;

  tmbuf = (struct tcpdirect_udma_mbuf *)calloc(1, sizeof(struct tcpdirect_udma_mbuf));
  if (!tmbuf) {
    exit(EXIT_FAILURE);
  }

  devfd = open("/dev/udmabuf", O_RDWR);
  if (devfd < 0) {
    printf("%s: [skip,no-udmabuf: Unable to access DMA buffer device file]\n",
           TEST_PREFIX);
    exit(70);
  }

  memfd = memfd_create("udmabuf-test", MFD_ALLOW_SEALING);
  if (memfd < 0) {
    printf("%s: [skip,no-memfd]\n", TEST_PREFIX);
    exit(72);
  }

  ret = fcntl(memfd, F_ADD_SEALS, F_SEAL_SHRINK);
  if (ret < 0) {
    printf("%s: [skip,fcntl-add-seals]\n", TEST_PREFIX);
    exit(73);
  }

  ret = ftruncate(memfd, size);
  if (ret == -1) {
    printf("%s: [FAIL,memfd-truncate]\n", TEST_PREFIX);
    exit(74);
  }

  memset(&create, 0, sizeof(create));

  create.memfd = memfd;
  create.offset = 0;
  create.size = size;
  printf("size=%lu\n", size);
  buf = ioctl(devfd, UDMABUF_CREATE, &create);
  if (buf < 0) {
    printf("%s: [FAIL, create udmabuf]\n", TEST_PREFIX);
    exit(75);
  }

  pages_create_info.dma_buf_fd = buf;
  pages_create_info.create_page_pool = is_client ? 0 : 1;

  /* TODO: hardcoded NIC pci address */
  // "0000:06:00.0"
  ret = sscanf(opts->tcpd_nic_pci_addr, "0000:%llx:%llx.%llx",
         &pages_create_info.pci_bdf[0],
         &pages_create_info.pci_bdf[1],
         &pages_create_info.pci_bdf[2]);

  if (ret != 3) {
    printf("%s: [FAIL, parse fail]\n", TEST_PREFIX);
    exit(76);
  }

  buf_pages = ioctl(buf, DMA_BUF_CREATE_PAGES, &pages_create_info);
  if (buf_pages < 0) {
    perror("ioctl DMA_BUF_CREATE_PAGES: [FAIL, create pages fail]\n");
    exit(77);
  }

  if (!is_client) {
    /* TODO hardcoded num_queues */
    int num_queues = 15;
    struct dma_buf_pages_bind_rx_queue bind_cmd;

    strcpy(bind_cmd.ifname, "eth1");
    bind_cmd.rxq_idx = num_queues;

    ret = ioctl(buf_pages, DMA_BUF_PAGES_BIND_RX, &bind_cmd);
    if (ret < 0) {
      printf("%s: [FAIL, bind fail queue=%d]\n", TEST_PREFIX,
            num_queues);
      exit(78);
    }
  }

  struct dma_buf_sync sync = { 0 };
  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_START;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  *f_mbuf = tmbuf;

  tmbuf->devfd = devfd;
  tmbuf->memfd = memfd;
  tmbuf->buf = buf;
  tmbuf->buf_pages = buf_pages;
  return 0;
}

int tcpdirect_udma_send(int socket, void *f_mbuf, size_t n, int flags) {
  int buf_pages, buf;
  struct iovec iov;
  struct msghdr *msg;
  struct cmsghdr *cmsg;
  char buf_dummy[n];
  char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 2)];
  struct tcpdirect_udma_mbuf *tmbuf;

  if (!f_mbuf) return -1;

  tmbuf = (struct tcpdirect_udma_mbuf *)f_mbuf;
  buf_pages = tmbuf->buf_pages;
  buf = tmbuf->buf;
  msg = &tmbuf->msg;

  struct dma_buf_sync sync = { 0 };
  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_START;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  char *buf_mem = NULL;
  buf_mem = (char *)mmap(NULL, n, PROT_READ | PROT_WRITE, MAP_SHARED, buf, 0);
  if (buf_mem == MAP_FAILED) {
    perror("mmap()");
    exit(1);
  }

  memcpy(buf_mem, buf_dummy, n);

  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_END;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  munmap(buf_mem, n);

  memset(msg, 0, sizeof(struct msghdr));
  // memset(cmsg, 0, sizeof(struct cmsghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n;

  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  cmsg = CMSG_FIRSTHDR(msg);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int*)CMSG_DATA(cmsg)) = buf_pages;
  ((int*)CMSG_DATA(cmsg))[1] = 0;

  ssize_t bytes_sent = sendmsg(socket, msg, MSG_ZEROCOPY);
  if (bytes_sent < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    perror("sendmsg() error: ");
    exit(EXIT_FAILURE);
  }

  if (bytes_sent == 0) {
    perror("sendmsg() sent 0 bytes. Something is wrong.\n");
    exit(EXIT_FAILURE);
  }

  return bytes_sent;
}

int tcpdirect_send(int socket, void *buf, size_t n, int flags) {
  int gpu_mem_fd_;
  struct iovec iov;
  struct msghdr *msg;
  struct cmsghdr *cmsg;
  char buf_dummy[n];
  char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 2)];
  struct tcpdirect_udma_mbuf *tmbuf;

  if (!buf) return -1;

  tmbuf = (struct tcpdirect_udma_mbuf *)buf;
  gpu_mem_fd_ = tmbuf->pages_fd;
  msg = &tmbuf->msg;

  memset(msg, 0, sizeof(struct msghdr));
  // memset(cmsg, 0, sizeof(struct cmsghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n;

  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  cmsg = CMSG_FIRSTHDR(msg);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int*)CMSG_DATA(cmsg)) = gpu_mem_fd_;

  ssize_t bytes_sent = sendmsg(socket, msg, MSG_ZEROCOPY | MSG_DONTWAIT);
  if (bytes_sent < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    perror("sendmsg() error: ");
    exit(EXIT_FAILURE);
  }

  if (bytes_sent == 0) {
    perror("sendmsg() sent 0 bytes. Something is wrong.\n");
    exit(EXIT_FAILURE);
  }

  return bytes_sent;
}

int tcpdirect_recv(int socket, void *f_mbuf, size_t n, int flags) {
  struct iovec iov;
  struct msghdr msg_local;
  struct msghdr *msg;
  struct tcpdirect_udma_mbuf *tmbuf;
  int buf, ret, client_fd;
  size_t total_received = 0;

  if (!f_mbuf) return -1;

  tmbuf = (struct tcpdirect_udma_mbuf *)f_mbuf;

  buf = tmbuf->buf;
  client_fd = socket;

  char buf_dummy[n];
  // char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 128)];
  char offsetbuf[CMSG_SPACE(sizeof(int) * 1000)];
  msg = &msg_local;

  memset(msg, 0, sizeof(struct msghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n;
  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  char *buf_mem = NULL;

  if (msg->msg_flags & MSG_CTRUNC) {
    printf("fatal, cmsg truncated, current msg_controllen\n");
 }

  ssize_t received = recvmsg(socket, msg, MSG_SOCK_DEVMEM | MSG_DONTWAIT);
  if (received < 0 && (errno == EAGAIN || errno == EWOULDBLOCK)) {
  } else if (received < 0) {
    printf("%s %d\n", __func__, __LINE__);
    return -1;
  } else if (received == 0) {
    printf("Client exited\n");
  }

  struct cmsghdr *cm = NULL;
  struct devmemvec *devmemvec = NULL;
  for (cm = CMSG_FIRSTHDR(msg); cm; cm = CMSG_NXTHDR(msg, cm)) {
    if (cm->cmsg_level != SOL_SOCKET ||
        (cm->cmsg_type != SCM_DEVMEM_OFFSET &&
          cm->cmsg_type != SCM_DEVMEM_HEADER)) {
      continue;
    }

    devmemvec = (struct devmemvec *)CMSG_DATA(cm);

    if (cm->cmsg_type == SCM_DEVMEM_HEADER) {
      // TODO: process data copied from skb's linear
      // buffer.
      fprintf(stderr, "\n\nSCM_DEVMEM_HEADER. devmemvec->frag_size=%u\n",
              devmemvec->frag_size);
      exit(1);
    }

    struct devmemtoken token = { devmemvec->frag_token, 1 };

    // struct dma_buf_sync sync = { 0 };
    // sync.flags = DMA_BUF_SYNC_READ | DMA_BUF_SYNC_START;
    // ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

    // buf_mem = (char *)mmap(NULL, n, PROT_READ | PROT_WRITE,
    //                MAP_SHARED, buf, 0);
    // if (buf_mem == MAP_FAILED) {
    //   perror("mmap()");
    //   exit(1);
    // }
    total_received += devmemvec->frag_size;
    printf("\n\nreceived frag_page=%u, in_page_offset=%u,"
            " frag_offset=%u, frag_size=%u, token=%u"
            " total_received=%lu\n",
            devmemvec->frag_offset >> PAGE_SHIFT,
            devmemvec->frag_offset % PAGE_SIZE,
            devmemvec->frag_offset, devmemvec->frag_size,
            devmemvec->frag_token,
            total_received);

    // sync.flags = DMA_BUF_SYNC_READ | DMA_BUF_SYNC_END;
    // ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

    ret = setsockopt(client_fd, SOL_SOCKET,
                      SO_DEVMEM_DONTNEED, &token,
                      sizeof(token));
    if (ret) {
      perror("DONTNEED failed");
      exit(1);
    }

    // munmap(buf_mem, n);
  }
  return total_received;
}

int cuda_flow_cleanup(void *f_mbuf) {
  struct tcpdirect_cuda_mbuf *t_mbuf = (struct tcpdirect_cuda_mbuf *)f_mbuf;

  close(t_mbuf->gpu_mem_fd_);
  close(t_mbuf->dma_buf_fd_);
  cudaFree(t_mbuf->gpu_tx_mem_);
  return 0;
}
#endif /* #ifdef WITH_TCPDIRECT */