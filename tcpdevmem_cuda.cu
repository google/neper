#include "hip/hip_runtime.h"
#define __iovec_defined 1

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <linux/uio.h>
#include <asm-generic/errno-base.h>
#include <asm-generic/socket.h>
#include <errno.h>
#include <fcntl.h>
#include <linux/if.h>
#include <stdlib.h>
#include <sys/ioctl.h>
#include <sys/mman.h>
#include <sys/socket.h>
#include <unistd.h>
#include <linux/dma-buf.h>

#include <memory>
#include <cstdint>
#include <cstdio>
#include <string>
#include <vector>

#if __cplusplus
extern "C" {
#endif

#include "common.h"
#include "tcpdevmem_cuda.h"
#include "tcpdevmem.h"
#include "logging.h"
#include "flow.h"
#include "thread.h"

#if __cplusplus
}
#endif

#define LAST_PRIME 111

#define MIN_RX_BUFFER_TOTAL_SIZE (1 << 28)
#define GPUMEM_ALIGNMENT (1UL << 21)
#define GPUMEM_MINSZ 0x400000

#define multiplier (1 << 16)

#define TEST_PREFIX "ncdevmem"
#define NUM_PAGES 16000

struct TcpdRxBlock {
  uint64_t gpu_offset;
  size_t size;
  uint64_t paddr;
};

/* Fills buf of size n with a repeating sequence of 1 to 111 inclusive
 */
void fill_tx_buffer(void *buf, size_t n) {
#define BUFSIZE 3996
  unsigned char src_buf[BUFSIZE];
  int ptr = 0, i = 0;

  while (i < BUFSIZE) {
    src_buf[i] = (i % LAST_PRIME) + 1;
    i++;
  }

  while (ptr*BUFSIZE + BUFSIZE < n) {
    hipMemcpy((char *)buf + ptr*BUFSIZE, &src_buf, BUFSIZE, hipMemcpyHostToDevice);
    ptr++;
  }

  i = ptr*BUFSIZE;
  while (i < n) {
    hipMemset((char *)buf + i, (i % LAST_PRIME) + 1, 1);
    i++;
  }
}

__global__ void scatter_copy_kernel(long3* scatter_list, uint8_t* dst,
                                    uint8_t* src) {
  int block_idx = blockIdx.x;
  long3 blk = scatter_list[block_idx];
  long dst_off = blk.x;
  long src_off = blk.y;
  long sz = blk.z;

  int thread_sz = sz / blockDim.x;
  int rem = sz % blockDim.x;
  bool extra = (threadIdx.x < rem);
  int thread_offset = sz / blockDim.x * threadIdx.x;
  thread_offset += (extra) ? threadIdx.x : rem;

  for (int i = 0; i < thread_sz; i++) {
    dst[dst_off + thread_offset + i] = src[src_off + thread_offset + i];
  }
  if (extra) {
    dst[dst_off + thread_offset + thread_sz] =
        src[src_off + thread_offset + thread_sz];
  }
}

void gather_rx_data(struct tcpdevmem_cuda_mbuf *tmbuf) {
  int ret;
  void *gpu_scatter_list_ = tmbuf->gpu_scatter_list_;
  std::vector<long3> *scattered_data_ = (std::vector<long3> *)tmbuf->scattered_data_;
  void *gpu_rx_mem_ = tmbuf->gpu_rx_mem_;
  void *rx_buff_ = tmbuf->gpu_gen_mem_;

  ret = hipMemcpyAsync(gpu_scatter_list_,
                        scattered_data_->data(),
                        scattered_data_->size() * sizeof(long3),
                        hipMemcpyHostToDevice);
  if (ret)
    return;

  scatter_copy_kernel<<<scattered_data_->size(), 256, 0>>>(
      (long3*)gpu_scatter_list_, (uint8_t*)gpu_rx_mem_, (uint8_t*)rx_buff_);
}

int get_gpumem_dmabuf_pages_fd(const std::string& gpu_pci_addr,
                               const std::string& nic_pci_addr, void* gpu_mem,
                               size_t gpu_mem_sz, int* dma_buf_fd, bool is_client) {
  int err, ret;

  hipMemGetHandleForAddressRange((void*)dma_buf_fd, (hipDeviceptr_t)gpu_mem,
                                gpu_mem_sz, hipMemRangeHandleTypeDmaBufFd,
                                0);

  if (*dma_buf_fd < 0) {
    perror("hipMemGetHandleForAddressRange() failed!: ");
    exit(EXIT_FAILURE);
  }

  printf("Registered dmabuf region 0x%p of %lu Bytes\n",
      gpu_mem, gpu_mem_sz);

  struct dma_buf_create_pages_info frags_create_info;
  frags_create_info.dma_buf_fd = *dma_buf_fd;
  frags_create_info.create_page_pool = is_client ? 0 : 1;

  uint16_t pci_bdf[3];
  ret = sscanf(nic_pci_addr.c_str(), "0000:%hx:%hx.%hx", &pci_bdf[0],
               &pci_bdf[1], &pci_bdf[2]);
  frags_create_info.pci_bdf[0] = pci_bdf[0];
  frags_create_info.pci_bdf[1] = pci_bdf[1];
  frags_create_info.pci_bdf[2] = pci_bdf[2];
  if (ret != 3) {
    err = -EINVAL;
    goto err_close_dmabuf;
  }

  ret = ioctl(*dma_buf_fd, DMA_BUF_CREATE_PAGES, &frags_create_info);
  if (ret < 0) {
    perror("Error getting dma_buf frags: ");
    err = -EIO;
    goto err_close_dmabuf;
  }
  return ret;

err_close_dmabuf:
  close(*dma_buf_fd);
  return err;
}

int tcpd_cuda_setup_alloc(const struct options *opts, void **f_mbuf, struct thread *t)
{
  bool is_client = opts->client;
  int ret;
  void *gpu_gen_mem_;
  int gpu_mem_fd_;
  int dma_buf_fd_;
  // int q_start = opts->queue_start;
  // int q_num = opts->queue_num;
  struct tcpdevmem_cuda_mbuf *tmbuf;
  const char *gpu_pci_addr = opts->tcpd_gpu_pci_addr;
  const char *nic_pci_addr = opts->tcpd_nic_pci_addr;
  size_t alloc_size = opts->tcpd_phys_len;

  tmbuf =
    (struct tcpdevmem_cuda_mbuf *)calloc(1, sizeof(struct tcpdevmem_cuda_mbuf));
  if (!tmbuf) {
    exit(EXIT_FAILURE);
  }

  if (alloc_size % GPUMEM_ALIGNMENT != 0) {
    alloc_size += GPUMEM_ALIGNMENT - (alloc_size % GPUMEM_ALIGNMENT);
  }

  hipMalloc(&gpu_gen_mem_, alloc_size);
  if (is_client && opts->tcpd_validate) {
          fill_tx_buffer(gpu_gen_mem_, alloc_size);
          hipDeviceSynchronize();
  }
  unsigned int flag = 1;
  hipPointerSetAttribute(&flag,
                        HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
                        (hipDeviceptr_t)gpu_gen_mem_);

  gpu_mem_fd_ = get_gpumem_dmabuf_pages_fd(gpu_pci_addr, nic_pci_addr,
                                           gpu_gen_mem_, alloc_size,
                                           &dma_buf_fd_, is_client);

  if (gpu_mem_fd_ < 0) {
    printf("get_gpumem_dmabuf_pages_fd() failed!: ");
    exit(71);
  }

  if (!is_client)
    install_flow_steering(opts, gpu_mem_fd_, t);

  *f_mbuf = tmbuf;
  tmbuf->gpu_mem_fd_ = gpu_mem_fd_;
  tmbuf->dma_buf_fd_ = dma_buf_fd_;
  tmbuf->gpu_gen_mem_ = gpu_gen_mem_;
  tmbuf->cpy_buffer = malloc(opts->buffer_size);
  tmbuf->vectors = new std::vector<devmemvec>();
  tmbuf->tokens = new std::vector<devmemtoken>();
  tmbuf->bytes_received = 0;
  tmbuf->bytes_sent = 0;

  hipMalloc(&tmbuf->gpu_rx_mem_, opts->buffer_size);
  hipMalloc(&tmbuf->gpu_scatter_list_, opts->buffer_size);
  tmbuf->rx_blks_ = new std::vector<TcpdRxBlock>();
  tmbuf->scattered_data_ = new std::vector<long3>();
  return 0;
}

int tcpd_send(int socket, void *buf, size_t n, int flags) {
  int gpu_mem_fd_;
  struct iovec iov;
  struct msghdr msg;
  struct cmsghdr *cmsg;
  char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 2)];
  struct tcpdevmem_cuda_mbuf *tmbuf;

  if (!buf) return -1;

  tmbuf = (struct tcpdevmem_cuda_mbuf *)buf;
  gpu_mem_fd_ = tmbuf->gpu_mem_fd_;

  memset(&msg, 0, sizeof(msg));
  // memset(cmsg, 0, sizeof(struct cmsghdr));

  iov.iov_base = NULL;
  iov.iov_len = n - tmbuf->bytes_sent;

  msg.msg_iov = &iov;
  msg.msg_iovlen = 1;

  msg.msg_control = offsetbuf;
  msg.msg_controllen = sizeof(offsetbuf);

  cmsg = CMSG_FIRSTHDR(&msg);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int*)CMSG_DATA(cmsg)) = gpu_mem_fd_;
  ((int *)CMSG_DATA(cmsg))[1] = (int)tmbuf->bytes_sent;

  ssize_t bytes_sent = sendmsg(socket, &msg, MSG_ZEROCOPY | MSG_DONTWAIT);
  if (bytes_sent < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    perror("sendmsg() error: ");
    exit(EXIT_FAILURE);
  }

  if (bytes_sent == 0) {
    perror("sendmsg() sent 0 bytes. Something is wrong.\n");
    exit(EXIT_FAILURE);
  }

  tmbuf->bytes_sent += bytes_sent;
  if (tmbuf->bytes_sent == n)
    tmbuf->bytes_sent = 0;

  return bytes_sent;
}

int tcpd_recv(int socket, void *f_mbuf, size_t n, int flags, struct thread *t) {
  struct iovec iov;
  struct msghdr msg_local;
  struct msghdr *msg;
  struct tcpdevmem_cuda_mbuf *tmbuf;
  int ret, client_fd;
  int buffer_size = n;
  size_t total_received = 0;
  unsigned char *cpy_buffer;
  const struct options *opts = t->opts;
  std::vector<devmemvec> *vectors;
  std::vector<devmemtoken> *tokens;
  std::vector<TcpdRxBlock> *rx_blks_;
  std::vector<long3> *scattered_data_;

  if (!f_mbuf) return -1;

  tmbuf = (struct tcpdevmem_cuda_mbuf *)f_mbuf;
  cpy_buffer = (unsigned char *)tmbuf->cpy_buffer;
  vectors = (std::vector<devmemvec> *)tmbuf->vectors;
  tokens = (std::vector<devmemtoken> *)tmbuf->tokens;
  rx_blks_ = (std::vector<TcpdRxBlock> *)tmbuf->rx_blks_;
  scattered_data_ = (std::vector<long3> *)tmbuf->scattered_data_;

  client_fd = socket;

  char buf_dummy[n];
  char offsetbuf[CMSG_SPACE(sizeof(int) * 1000)];
  msg = &msg_local;

  memset(msg, 0, sizeof(struct msghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n - tmbuf->bytes_received;
  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  rx_blks_->clear();

  ssize_t received = recvmsg(socket, msg, MSG_SOCK_DEVMEM | MSG_DONTWAIT);
  if (received < 0 && (errno == EAGAIN || errno == EWOULDBLOCK)) {
    printf("%s %d: recvmsg returned < 0\n", __func__, __LINE__);
    return -1;
  } else if (received < 0) {
    printf("%s %d\n", __func__, __LINE__);
    return -1;
  } else if (received == 0) {
    printf("Client exited\n");
    return -1;
  }

  if (msg->msg_flags & MSG_CTRUNC) {
    LOG_ERROR(t->cb, "fatal, cmsg truncated, current msg_controllen");
  }

  struct cmsghdr *cm = NULL;
  struct devmemvec *devmemvec = NULL;
  for (cm = CMSG_FIRSTHDR(msg); cm; cm = CMSG_NXTHDR(msg, cm)) {
    if (cm->cmsg_level != SOL_SOCKET ||
        (cm->cmsg_type != SCM_DEVMEM_OFFSET &&
          cm->cmsg_type != SCM_DEVMEM_HEADER)) {
      continue;
    }

    devmemvec = (struct devmemvec *)CMSG_DATA(cm);

    if (cm->cmsg_type == SCM_DEVMEM_HEADER) {
      // TODO: process data copied from skb's linear
      // buffer.
      fprintf(stderr, "\n\nSCM_DEVMEM_HEADER. devmemvec->frag_size=%u\n",
              devmemvec->frag_size);
      exit(1);
    }

    struct devmemtoken token = { devmemvec->frag_token, 1 };
    struct TcpdRxBlock blk;

    if (devmemvec->frag_size > PAGE_SIZE)
      continue;

    blk.gpu_offset = (uint64_t)devmemvec->frag_offset;
    blk.size = devmemvec->frag_size;
    rx_blks_->emplace_back(blk);

    total_received += devmemvec->frag_size;

    vectors->emplace_back(*devmemvec);
    tokens->push_back(token);
  }

  size_t dst_offset = tmbuf->bytes_received;
  for (int i = 0; i < rx_blks_->size(); i++) {
    struct TcpdRxBlock blk = rx_blks_->at(i);
    size_t off = (size_t)blk.gpu_offset;
    scattered_data_->emplace_back(
        make_long3((long)dst_offset, (long)off, (long)blk.size));

    dst_offset += blk.size;
  }
  tmbuf->bytes_received += received;

  /* Once we've received fragments totaling buffer_size, we can copy from the
   * CUDA buffer to a user-space buffer, and free the fragments in the CUDA
   * buffer.
  */
  if (tmbuf->bytes_received == buffer_size) {
    if (opts->tcpd_rx_cpy) {
      gather_rx_data(tmbuf);
      hipDeviceSynchronize();
    }
    /* There is a performance impact when we hipMemcpy from the CUDA buffer to
     * the userspace buffer, so it's gated by a flag
     */
    if (opts->tcpd_validate) {
      for (int idx = 0; idx < vectors->size(); idx++) {
        struct devmemvec vec = (*vectors)[idx];
        struct devmemtoken token = (*tokens)[idx];

        /* copy each fragment to the cpy_buffer in order, i.e.
         * 1st fragment will occuply bytes [0-4095], 2nd fragment will
         * occupy bytes [4096-8191], etc.
         */
        hipMemcpy(cpy_buffer + (vec.frag_token - 1) * PAGE_SIZE,
                   (char *)tmbuf->gpu_gen_mem_ + vec.frag_offset,
                   vec.frag_size,
                   hipMemcpyDeviceToHost);
      }

      /* Ensure the sequence is what we expect:
       * a repeating sequence of 1 to LAST_PRIME inclusive
       */
      hipDeviceSynchronize();
      int i = 0;
      int expected_val;
      while (i < buffer_size) {
        expected_val = (i % LAST_PRIME) + 1;
        if (cpy_buffer[i] != expected_val) {
          LOG_WARN(t->cb,
                   "Thread %i - incorrect byte %i, expected %i, got %i",
                   t->index,
                   i,
                   expected_val,
                   cpy_buffer[i]);
          break;
        }
        i++;
      }
    }

    ret = setsockopt(client_fd, SOL_SOCKET,
                      SO_DEVMEM_DONTNEED, tokens->data(),
                      tokens->size() * sizeof(devmemtoken));
    if (ret) {
      perror("DONTNEED failed");
      exit(1);
    }
    vectors->clear();
    tokens->clear();
    rx_blks_->clear();
    scattered_data_->clear();
    tmbuf->bytes_received = 0;
  }
  return total_received;
}

int cuda_flow_cleanup(void *f_mbuf) {
  struct tcpdevmem_cuda_mbuf *t_mbuf = (struct tcpdevmem_cuda_mbuf *)f_mbuf;
  close(t_mbuf->gpu_mem_fd_);
  close(t_mbuf->dma_buf_fd_);
  hipFree(t_mbuf->gpu_gen_mem_);
  free(t_mbuf->cpy_buffer);
  free(t_mbuf->tokens);
  free(t_mbuf->vectors);

  hipFree(t_mbuf->gpu_rx_mem_);
  hipFree(t_mbuf->gpu_scatter_list_);
  free(t_mbuf->rx_blks_);
  free(t_mbuf->scattered_data_);
  return 0;
}
